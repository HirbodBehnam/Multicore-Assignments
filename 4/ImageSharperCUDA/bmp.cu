#include "hip/hip_runtime.h"
#include <iostream>
#include "bmp.cuh"

#define DATA_OFFSET_OFFSET 0x000A
#define WIDTH_OFFSET 0x0012
#define HEIGHT_OFFSET 0x0016
#define BITS_PER_PIXEL_OFFSET 0x001C
#define HEADER_SIZE 14
#define INFO_HEADER_SIZE 40
#define NO_COMPRESION 0
#define MAX_NUMBER_OF_COLORS 0
#define ALL_COLORS_REQUIRED 0

//***Inputs*****
//fileName: The name of the file to open 
//***Outputs****
//pixels: A pointer to a byte array. This will contain the pixel data
//width: An int pointer to store the width of the image in pixels
//height: An int pointer to store the height of the image in pixels
//bytesPerPixel: An int pointer to store the number of bytes per pixel that are used in the image
void ReadImage(const char *fileName, uint8_t **pixels, uint32_t *width, uint32_t *height, uint32_t *bytesPerPixel) {
    //Open the file for reading in binary mode
    FILE *imageFile = fopen(fileName, "rb");
    //Read data offset
    uint32_t dataOffset;
    fseek(imageFile, DATA_OFFSET_OFFSET, SEEK_SET);
    fread(&dataOffset, 4, 1, imageFile);
    //Read width
    fseek(imageFile, WIDTH_OFFSET, SEEK_SET);
    fread(width, 4, 1, imageFile);
    //Read height
    fseek(imageFile, HEIGHT_OFFSET, SEEK_SET);
    fread(height, 4, 1, imageFile);
    //Read bits per pixel
    int16_t bitsPerPixel;
    fseek(imageFile, BITS_PER_PIXEL_OFFSET, SEEK_SET);
    fread(&bitsPerPixel, 2, 1, imageFile);
    //Allocate a pixel array
    *bytesPerPixel = ((uint32_t) bitsPerPixel) / 8;

    //Rows are stored bottom-up
    //Each row is padded to be a multiple of 4 bytes.
    //We calculate the padded row size in bytes
    int paddedRowSize = (int) (4 * ceil((float) (*width) / 4.0f)) * (*bytesPerPixel);
    //We are not interested in the padded bytes, so we allocate memory just for
    //the pixel data
    int unpaddedRowSize = (*width) * (*bytesPerPixel);
    //Total size of the pixel data in bytes
    int totalSize = unpaddedRowSize * (*height);
    *pixels = (uint8_t *) malloc(totalSize);
    //Read the pixel data Row by Row.
    //Data is padded and stored bottom-up
    int i = 0;
    //point to the last row of our pixel array (unpadded)
    uint8_t *currentRowPointer = *pixels + ((*height - 1) * unpaddedRowSize);
    for (i = 0; i < *height; i++) {
        //put file cursor in the next row from top to bottom
        fseek(imageFile, dataOffset + (i * paddedRowSize), SEEK_SET);
        //read only unpaddedRowSize bytes (we can ignore the padding bytes)
        fread(currentRowPointer, 1, unpaddedRowSize, imageFile);
        //point to the next row (from bottom to top)
        currentRowPointer -= unpaddedRowSize;
    }

    fclose(imageFile);
}

//***Inputs*****
//fileName: The name of the file to save 
//pixels: Pointer to the pixel data array
//width: The width of the image in pixels
//height: The height of the image in pixels
//bytesPerPixel: The number of bytes per pixel that are used in the image
void WriteImage(const char *fileName, uint8_t *pixels, uint32_t width, uint32_t height, uint32_t bytesPerPixel) {
    //Open file in binary mode
    FILE *outputFile = fopen(fileName, "wb");
    //*****HEADER************//
    //write signature
    const char *BM = "BM";
    fwrite(&BM[0], 1, 1, outputFile);
    fwrite(&BM[1], 1, 1, outputFile);
    //Write file size considering padded bytes
    int paddedRowSize = (int) (4 * ceil((float) width / 4.0f)) * bytesPerPixel;
    uint32_t fileSize = paddedRowSize * height + HEADER_SIZE + INFO_HEADER_SIZE;
    fwrite(&fileSize, 4, 1, outputFile);
    //Write reserved
    uint32_t reserved = 0x0000;
    fwrite(&reserved, 4, 1, outputFile);
    //Write data offset
    uint32_t dataOffset = HEADER_SIZE + INFO_HEADER_SIZE;
    fwrite(&dataOffset, 4, 1, outputFile);

    //*******INFO*HEADER******//
    //Write size
    uint32_t infoHeaderSize = INFO_HEADER_SIZE;
    fwrite(&infoHeaderSize, 4, 1, outputFile);
    //Write width and height
    fwrite(&width, 4, 1, outputFile);
    fwrite(&height, 4, 1, outputFile);
    //Write planes
    int16_t planes = 1; //always 1
    fwrite(&planes, 2, 1, outputFile);
    //write bits per pixel
    int16_t bitsPerPixel = bytesPerPixel * 8;
    fwrite(&bitsPerPixel, 2, 1, outputFile);
    //write compression
    uint32_t compression = NO_COMPRESION;
    fwrite(&compression, 4, 1, outputFile);
    //write image size (in bytes)
    uint32_t imageSize = width * height * bytesPerPixel;
    fwrite(&imageSize, 4, 1, outputFile);
    //write resolution (in pixels per meter)
    uint32_t resolutionX = 11811; //300 dpi
    uint32_t resolutionY = 11811; //300 dpi
    fwrite(&resolutionX, 4, 1, outputFile);
    fwrite(&resolutionY, 4, 1, outputFile);
    //write colors used
    uint32_t colorsUsed = MAX_NUMBER_OF_COLORS;
    fwrite(&colorsUsed, 4, 1, outputFile);
    //Write important colors
    uint32_t importantColors = ALL_COLORS_REQUIRED;
    fwrite(&importantColors, 4, 1, outputFile);
    //write data
    int i = 0;
    int unpaddedRowSize = width * bytesPerPixel;
    for (i = 0; i < height; i++) {
        //start writing from the beginning of last row in the pixel array
        int pixelOffset = ((height - i) - 1) * unpaddedRowSize;
        fwrite(&pixels[pixelOffset], 1, paddedRowSize, outputFile);
    }
    fclose(outputFile);
}
