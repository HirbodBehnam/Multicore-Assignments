#include <opencv2/opencv.hpp>
#include "saver.cuh"

void save_grayscale_image(const char *output_path, const uint8_t *image_buffer, int width, int height) {
    cv::Mat image(width, height, CV_8UC1, cv::Scalar(0));
    hipMemcpy(image.data, image_buffer, width * height, hipMemcpyDeviceToHost);
    cv::imwrite(output_path, image);
}