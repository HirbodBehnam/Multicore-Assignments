#include <iostream>
#include "reader.cuh"
#include "brighter.cuh"
#include "saver.cuh"
#include "sobel.cuh"

int main(int argc, char **argv) {
    // Check arguments
    if (argc < 2) {
        std::cout << "Program usage:" << std::endl << argv[0]
                  << " INPUT_NAME [BRIGHTNESS_ALPHA] [BRIGHTNESS_BETA] [THRESHOLD]"
                  << std::endl;
        exit(1);
    }
    float alpha = argc > 2 ? strtof(argv[2], nullptr) : 1.0f;
    float beta = argc > 3 ? strtof(argv[3], nullptr) : 0.0f;
    auto threshold = static_cast<int16_t>(argc > 4 ? strtol(argv[4], nullptr, 10) : 70);
    std::cout << "Running with " << alpha << " as alpha and " << beta << "as beta and " << threshold << " as threshold"
              << std::endl;
    // Read the image
    std::cout << "Reading input..." << std::endl;
    uint8_t *grayscale_image;
    int width, height;
    read_to_grayscale(argv[1], &grayscale_image, &width, &height);
    save_grayscale_image_gpu("grayscale.png", grayscale_image, width, height);
    // Brighten it
    std::cout << "Brightening input..." << std::endl;
    brighter(grayscale_image, width * height, alpha, beta);
    save_grayscale_image_gpu("brighten.png", grayscale_image, width, height);
    // Edge detect it
    uint8_t *Gx, *Gy, *G;
    hipMalloc(&Gx, width * height);
    hipMalloc(&Gy, width * height);
    hipMalloc(&G, width * height);
    std::cout << "Edge detecting..." << std::endl;
    sobel_edge_detection(grayscale_image, width, height, threshold, Gx, Gy, G);
    // Save images
    save_grayscale_image_gpu("Gx.png", Gx, width, height);
    save_grayscale_image_gpu("Gy.png", Gy, width, height);
    save_grayscale_image_gpu("G.png", G, width, height);
    // Cleanup
    hipFree(G);
    hipFree(Gy);
    hipFree(Gy);
    hipFree(grayscale_image);
    return 0;
}
